#include <vector>
#include <string>
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>



#ifdef _WIN32

#include <sys/timeb.h>
#include <time.h>
#include <float.h>
#include <windows.h>

double gettime() { // granularity about 50 microsecs on my machine
	static LARGE_INTEGER freq, start;
	LARGE_INTEGER count;
	if (!QueryPerformanceCounter(&count))
		// FatalError("QueryPerformanceCounter");
		fprintf(stderr, "QueryPerformanceCounter");
	if (!freq.QuadPart) { // one time initialization
		if (!QueryPerformanceFrequency(&freq))
			//FatalError("QueryPerformanceFrequency");
			fprintf(stderr, "QueryPerformanceCounter");
		start = count;
	}
	return (double)(count.QuadPart - start.QuadPart) / freq.QuadPart;
}


int     opterr = 1,             /* if error message should be printed */
optind = 1,             /* index into parent argv vector */
optopt,                 /* character checked for validity */
optreset;               /* reset getopt */
char    *optarg;                /* argument associated with option */

#define BADCH   (int)'?'
#define BADARG  (int)':'
#define EMSG    ""

/*
* getopt --
*      Parse argc/argv argument vector.
*/
int
getopt(int nargc, char * const nargv[], const char *ostr)
{
	static char *place = EMSG;              /* option letter processing */
	const char *oli;                        /* option letter list index */

	if (optreset || !*place) {              /* update scanning pointer */
		optreset = 0;
		if (optind >= nargc || *(place = nargv[optind]) != '-') {
			place = EMSG;
			return (-1);
		}
		if (place[1] && *++place == '-') {      /* found "--" */
			++optind;
			place = EMSG;
			return (-1);
		}
	}                                       /* option letter okay? */
	if ((optopt = (int)*place++) == (int)':' ||
		!(oli = strchr(ostr, optopt))) {
		/*
		* if the user didn't specify '-' as an option,
		* assume it means -1.
		*/
		if (optopt == (int)'-')
			return (-1);
		if (!*place)
			++optind;
		if (opterr && *ostr != ':')
			(void)printf("illegal option -- %c\n", optopt);
		return (BADCH);
	}
	if (*++oli != ':') {                    /* don't need argument */
		optarg = NULL;
		if (!*place)
			++optind;
	}
	else {                                  /* need an argument */
		if (*place)                     /* no white space */
			optarg = place;
		else if (nargc <= ++optind) {   /* no arg */
			place = EMSG;
			if (*ostr == ':')
				return (BADARG);
			if (opterr)
				(void)printf("option requires an argument -- %c\n", optopt);
			return (BADCH);
		}
		else                            /* white space */
			optarg = nargv[optind];
		place = EMSG;
		++optind;
	}
	return (optopt);                        /* dump back option letter */
}
#else

#include <sys/time.h>
#include <getopt.h>
#define _isnan(a) isnan(a)

double gettime() { // returns 0 seconds first time called
	static struct timeval t0;
	struct timeval tv;
	gettimeofday(&tv, 0);
	if (!t0.tv_sec)
		t0 = tv;
	return tv.tv_sec - t0.tv_sec + (tv.tv_usec - t0.tv_usec) / 1000000.;
}

/*
double gettime() {
timespec ts;
clock_gettime(CLOCK_REALTIME, &ts);
return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}
/**/

#endif



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, 
int line, bool abort = false)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(1);
	}
}
