#include "hip/hip_runtime.h"

//Fills the vector that counts the occurrences of each feature/item (document frequency - DF) 
void build_occurrences(int* occurrences){

	vector< vector<short> >& unlabeled_partition = UNLABELED;
	memset(occurrences, 0, sizeof(int) * MAX_ITEMS);

#pragma omp parallel for	
	for (int i = 0; i < unlabeled_partition.size(); i++){
		for (int j = 1; j < unlabeled_partition[i].size(); j++){
			int x = unlabeled_partition[i][j];
#pragma omp atomic
			occurrences[x]++;
		}
	}
}

//Fills the vector that store the sum of the DFs of the items that compose the document, for all documents
void docs_DF_sum(vector< vector<short> >& unlabeled_partition, int* occurrences, vector<int>& doc_occurs){

	doc_occurs.resize(unlabeled_partition.size());

#pragma omp parallel for
	for (int i = 0; i < unlabeled_partition.size(); i++){

		int cnt = 0;

		for (int j = 1; j < unlabeled_partition[i].size(); j++){
			cnt += occurrences[unlabeled_partition[i][j]];
		}

		doc_occurs[i] = cnt;
	}
}

//First chooses the document that has the greatest DF sum. On a tie, choose the one with greater id
int find_most_representative(vector< vector<short> >& unlabeled_partition, vector<int>& doc_occurs){

	int max = -1, idx = 0;

	for (int i = 0; i < unlabeled_partition.size(); i++){

		int cnt = doc_occurs[i];

		if (max < cnt){
			max = cnt;
			idx = i;
		}
		else if (max == cnt && idx < i)
			idx = i;
	}

	return idx;
}

//Merge the reduced sets produced from each partition, with a set union operation
void merge(vector< vector<int> >& reduced_trs_ids, int partitions){

	fprintf(stderr, "Merging reduced sets\n");
	set<int> reduced_uniq;

	//merge with their ids and a set container
	for (int p = 0; p < partitions; p++){
		for (int i = 0; i < reduced_trs_ids[p].size(); i++)
			reduced_uniq.insert(reduced_trs_ids[p][i]);
	}

	fprintf(stderr, "Reduced set has %u lines.\n", reduced_uniq.size());

	//Then print the reduced set according to the original data that was read
	for (set<int>::iterator it = reduced_uniq.begin(); it != reduced_uniq.end(); it++){

		printf("%d CLASS=%d ", *it + 1, UNLABELED[*it][0]);
		for (int i = 1; i < UNLABELED[*it].size(); i++)
			printf("%s ", SYMBOL_TABLE[UNLABELED[*it][i]].data());
		putchar('\n');
	}
}

//Creates the partitions by using the ordered features file and the read data.
void partitioner(vector< vector< vector<short> > >& unlabeled_partitions, char *file_features){

	vector<short> ordered_features;
	read_ordered_features(ordered_features, file_features);
	int partitions = (int)unlabeled_partitions.size();

	for (int i = 0; i < UNLABELED.size(); i++){

		for (int j = 0; j < partitions; j++){
			unlabeled_partitions[j].push_back(vector<short>());
			unlabeled_partitions[j][i].push_back(UNLABELED[i][0]); //class
		}

		int k = 0;
		//Assign with a round-robin manner
		for (int j = 0; j < ordered_features.size(); j++){

			unlabeled_partitions[k % partitions][i].push_back(UNLABELED[i][ordered_features[j]]);
			k++;
		}
	}
}

